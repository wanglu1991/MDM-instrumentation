#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <string>
#include <map>
#include <set>
/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static __managed__ ChannelDev channel_dev_inst;
static ChannelHost channel_host;
static ChannelHost channel_host_inst;
/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;
int cache_line_size = 128;
/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;
int kernel_id=0;
/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

/* information collected in the instrumentation function */
typedef struct {
    int cta_id_x;
    int cta_id_y;
    int cta_id_z;
    int warp_id;
    int opcode_id;
    int offset;
    int RoW;
    int sm_id;
    uint64_t addrs[32];
} mem_access_t;

typedef struct
{ int warp_id;
  int opcode_id;
  int offset;
  int des_reg;
  int source_reg_1;
  int source_reg_2;
}instruction_t;







/* instrumentation function that we want to inject, please note the use of
 * 1. "extern "C" __device__ __noinline__" to prevent code elimination by the
 * compiler.
 * 2. NVBIT_EXPORT_FUNC(count_instrs) to notify nvbit the name of the function
 * we want to inject. This name must match exactly the function name */
extern "C" __device__ __noinline__ void dep_instrs(int predicate,
                                                     int count_warp_level,int offset,int opcode_id, int des_reg=10000,int reg_2=10000,int reg_3=10000) {
    /* all the active threads will compute the active mask */
    const int active_mask = __ballot(1);
    /* compute the predicate mask */
    const int predicate_mask = __ballot(predicate);
    /* each thread will get a lane id (get_lane_id is in utils/utils.h) */
    const int laneid = get_laneid();
    /* get the id of the first active thread */
    const int first_laneid = __ffs(active_mask) - 1;
    /* count all the active thread */
    const int num_threads = __popc(predicate_mask);
    const int warp_id =get_global_warp_id();
    instruction_t ta;
    ta.warp_id = warp_id;
    ta.offset = offset;
    ta.opcode_id = opcode_id;
    ta.des_reg = des_reg;
    if(reg_2<1000)
    	ta.source_reg_1=reg_2;
    if(reg_3<1000)
    	ta.source_reg_2=reg_3;

    /* only the first active thread will perform the atomic */
    // inst->print();
    if (first_laneid == laneid)
            {
        if (count_warp_level)
            {
            /* num threads can be zero when accounting for predicates off */
            if (num_threads > 0)
            {atomicAdd((unsigned long long *)&counter, 1);
             channel_dev_inst.push(&ta, sizeof(instruction_t));
            if(warp_id==1)
            //FILE * f =fopen("./instruction_trace.txt","a");
            //if(f!=NULL)
            printf("%d,%d,%d,%d,%d,%d\n",warp_id,offset,opcode_id,des_reg,reg_2,reg_3);
            //fclose(f);
            }
            else
            {
            atomicAdd((unsigned long long *)&counter, num_threads);
            }
            }
            }
    }
NVBIT_EXPORT_FUNC(dep_instrs);

/* Instrumentation function that we want to inject, please note the use of
 * 1. extern "C" __device__ __noinline__
 *    To prevent "dead"-code elimination by the compiler.
 * 2. NVBIT_EXPORT_FUNC(dev_func)
 *    To notify nvbit the name of the function we want to inject.
 *    This name must match exactly the function name.
 */
extern "C" __device__ __noinline__ void instrument_mem(int pred, int opcode_id,
                                                       uint32_t reg_high,
                                                       uint32_t reg_low,
                                                       int32_t imm,int offset,int RoW) {
    if (!pred) {
        return;
    }

    int64_t base_addr = (((uint64_t)reg_high) << 32) | ((uint64_t)reg_low);
    uint64_t addr = base_addr + imm;

    int active_mask = __ballot(1);
    const int laneid = get_laneid();
    const int first_laneid = __ffs(active_mask) - 1;

    mem_access_t ma;
    /* collect memory address information */
    for (int i = 0; i < 32; i++) {
        ma.addrs[i] = __shfl(addr, i);
    }

    int4 cta = get_ctaid();
    ma.cta_id_x = cta.x;
    ma.cta_id_y = cta.y;
    ma.cta_id_z = cta.z;
    ma.warp_id = get_global_warp_id();
    ma.sm_id = get_smid();
    ma.opcode_id = opcode_id;
    ma.offset  = offset;
    ma.RoW =RoW;
    /* first active lane pushes information on the channel */
    if (first_laneid == laneid) {
        channel_dev.push(&ma, sizeof(mem_access_t));
    }
}
NVBIT_EXPORT_FUNC(instrument_mem);

void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
}

/* instrument each memory instruction adding a call to the above instrumentation
 * function */
void nvbit_at_function_first_load(hipCtx_t ctx, hipFunction_t f) {
    const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
    if (verbose) {
        printf("Inspecting function %s at address 0x%lx\n",
               nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
    }

    uint32_t cnt = 0;
    /* iterate on all the static instructions in the function */
    for (auto instr : instrs) {
        if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
            instr->getMemOpType() == Instr::NONE) {
            cnt++;
            continue;
        }
        if (verbose) {
            instr->printDecoded();
        }

        if (opcode_to_id_map.find(instr->getOpcode()) ==
            opcode_to_id_map.end()) {
            int opcode_id = opcode_to_id_map.size();
            opcode_to_id_map[instr->getOpcode()] = opcode_id;
            printf("OPCODE %s MAPS TO ID %d\n",i->getOpcode(),opcode_id);
            id_to_opcode_map[opcode_id] = std::string(instr->getOpcode());
        }

        int opcode_id = opcode_to_id_map[instr->getOpcode()];
        int offset = instr->getOffset();

        /* instrument for instruction trace */

        {
        nvbit_insert_call(i, "dep_instrs", IPOINT_BEFORE);
                  if (exclude_pred_off) {
                      /* pass predicate value */
                      nvbit_add_call_arg_pred_val(i);
                  } else {
                      /* pass always true */
                      nvbit_add_call_arg_const_val32(i, 1);
                  }

        /* add count warps option */
        nvbit_add_call_arg_const_val32(i, count_warp_level);
        /* add instruction pc */
        nvbit_add_call_arg_const_val32(i,offset);
        /* add opcode */
        nvbit_add_call_arg_const_val32(i,opcode_id);
        //if(!i->isStore())
                 // {
        for (int j=0;j<i->getNumOperands();j++)
         {const Instr::operand_t * op=i->getOperand(j);/*get each operand*/
          if((op->type==Instr::REG))
          nvbit_add_call_arg_const_val32(i,op->value[0]);/* get register_id*/
          else
          {
             if(j==0)
             nvbit_add_call_arg_const_val32(i,10000);
           }
         }
        }

        /* instrument for memory trace */
        int RoW = 0;
        if(instr->isLoad())
        RoW=1;
        /* iterate on the operands */
        for (int i = 0; i < instr->getNumOperands(); i++) {
            /* get the operand "i" */
            const Instr::operand_t *op = instr->getOperand(i);

            if ((op->type == Instr::MREF)&&(instr->getMemOpType()==Instr::GLOBAL)) {
                /* insert call to the instrumentation function with its
                    * arguments */
                nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
                nvbit_add_call_arg_pred_val(instr);
                nvbit_add_call_arg_const_val32(instr, opcode_id);
                if (instr->isExtended()) {
                    nvbit_add_call_arg_reg_val(instr, (int)op->value[0] + 1);
                } else {
                    nvbit_add_call_arg_reg_val(instr, (int)Instr::RZ);
                }
                nvbit_add_call_arg_reg_val(instr, (int)op->value[0]);
                nvbit_add_call_arg_const_val32(instr, (int)op->value[1]);
                nvbit_add_call_arg_const_val32(instr,offset);
                nvbit_add_call_arg_const_val32(instr,RoW);
            }
        }
        cnt++;
    }
}

__global__ void flush_channel() {
    /* push memory access with negative cta id to communicate the kernel is
     * completed */
    mem_access_t ma;
    ma.cta_id_x = -1;
    channel_dev.push(&ma, sizeof(mem_access_t));

    /* flush channel */
    channel_dev.flush();

    instruction_t ta;
    ta.warp_id=-1;
    channel_dev_inst.push(&ta, sizeof(instruction_t));

       /* flush channel */
    channel_dev_inst.flush();
}



/*
__global__ void flush_channel() {
    //push memory access with negative cta id to communicate the kernel is completed

}
*/
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    if (skip_flag) return;

    if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel) {
        cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

        if (!is_exit) {
            int nregs;
            CUDA_SAFECALL(
                hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

            int shmem_static_nbytes;
            CUDA_SAFECALL(hipFuncGetAttribute(&shmem_static_nbytes,
                                          HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES,
                                          p->f));

            
            printf(
                "Kernel %s - grid size %d,%d,%d - block size %d,%d,%d - nregs "
                "%d - shmem %d - cuda stream id %ld\n",
                nvbit_get_func_name(ctx, p->f), p->gridDimX, p->gridDimY,
                p->gridDimZ, p->blockDimX, p->blockDimY, p->blockDimZ, nregs,
                shmem_static_nbytes + p->sharedMemBytes, (uint64_t)p->hStream);
            
            recv_thread_receiving = true;

        } else {
            kernel_id++;
            /* make sure current kernel is completed */
            hipDeviceSynchronize();
            assert(hipGetLastError() == hipSuccess);

            /* make sure we prevent re-entry on the nvbit_callback when issuing
             * the flush_channel kernel */
            skip_flag = true;

            /* issue flush of channel so we are sure all the memory accesses
             * have been pushed */
            flush_channel<<<1, 1>>>();
            hipDeviceSynchronize();
            assert(hipGetLastError() == hipSuccess);

            /* unset the skip flag */
            skip_flag = false;

            /* wait here until the receiving thread has not finished with the
             * current kernel */
            while (recv_thread_receiving) {
                pthread_yield();
            }
        }
    }
}

void *recv_thread_fun(void *) {
    char *recv_buffer = (char *)malloc(CHANNEL_SIZE);
    std::map<int, std::vector<mem_access_t *>> per_warp_mem_trace;

    while (recv_thread_started) {
        uint32_t num_recv_bytes = 0;
        if (recv_thread_receiving &&
            (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
                0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {
                mem_access_t *ma =
                    (mem_access_t *)&recv_buffer[num_processed_bytes];

                /* when we get this cta_id_x it means the kernel has completed
                 */
                if (ma->cta_id_x == -1) {
                    recv_thread_receiving = false;
                    break;
                }
                int warp_id =ma->warp_id;
                per_warp_mem_trace[warp_id].push_back(ma);
                char fn[100];
                snprintf(fn,sizeof(fn),"./mem_trace_%d.txt",kernel_id);
                FILE * f = fopen(fn,"a");
                if(f!=NULL)
                {fprintf(f,"%d,%d,%d,%d,",ma->warp_id,ma->sm_id,ma->offset,ma->RoW);
                 std::set<uint64_t> coalesced_addr;
                 for(int i=0;i<32;i++)
                 {uint64_t cache_line_addr=ma->addrs[i]/cache_line_size;
                  coalesced_addr.insert(cache_line_addr);
                 }
                 for(std::set<uint64_t>:: iterator addr = coalesced_addr.begin(); addr!=coalesced_addr.end();++addr)
                 fprintf(f,"%lld,",*addr);
                 fprintf(f,"\n");
                }
                fclose(f);
               /*
                printf("CTA %d,%d,%d - warp %d - %s - ", ma->cta_id_x,
                       ma->cta_id_y, ma->cta_id_z, ma->warp_id,
                       id_to_opcode_map[ma->opcode_id].c_str());
                for (int i = 0; i < 32; i++) {
                    printf("0x%016lx ", ma->addrs[i]);
            
                }
                printf("\n");
                */
                num_processed_bytes += sizeof(mem_access_t);
            }
        }
    }

    while (recv_thread_started) {
          uint32_t num_recv_bytes = 0;
          if (recv_thread_receiving &&
              (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
                  0) {
              uint32_t num_processed_bytes = 0;
              while (num_processed_bytes < num_recv_bytes) {
                  mem_access_t *ma =
                      (mem_access_t *)&recv_buffer[num_processed_bytes];

                  /* when we get this cta_id_x it means the kernel has completed
                   */
                  if (ma->cta_id_x == -1) {
                      recv_thread_receiving = false;
                      break;
                  }
                  int warp_id =ma->warp_id;
                  per_warp_mem_trace[warp_id].push_back(ma);
                  char fn[100];
                  snprintf(fn,sizeof(fn),"./mem_trace_%d.txt",kernel_id);
                  FILE * f = fopen(fn,"a");
                  if(f!=NULL)
                  {fprintf(f,"%d,%d,%d,%d,",ma->warp_id,ma->sm_id,ma->offset,ma->RoW);
                   std::set<uint64_t> coalesced_addr;
                   for(int i=0;i<32;i++)
                   {uint64_t cache_line_addr=ma->addrs[i]/cache_line_size;
                    coalesced_addr.insert(cache_line_addr);
                   }
                   for(std::set<uint64_t>:: iterator addr = coalesced_addr.begin(); addr!=coalesced_addr.end();++addr)
                   fprintf(f,"%lld,",*addr);
                   fprintf(f,"\n");
                  }
                  fclose(f);
                 /*
                  printf("CTA %d,%d,%d - warp %d - %s - ", ma->cta_id_x,
                         ma->cta_id_y, ma->cta_id_z, ma->warp_id,
                         id_to_opcode_map[ma->opcode_id].c_str());
                  for (int i = 0; i < 32; i++) {
                      printf("0x%016lx ", ma->addrs[i]);

                  }
                  printf("\n");
                  */
                  num_processed_bytes += sizeof(mem_access_t);
              }
          }
      }



    /*
    for(std::map<int,std::vector<mem_access_t *>>::iterator it=per_warp_mem_trace.begin(); it!=per_warp_mem_trace.end();++it)
    {std::vector<mem_access_t *> trace = it->second;
     FILE * f =fopen("./mem_trace.txt","a");
     if(f!=NULL)
     {
     for (int i =0; i< trace.size();i++)
     {fprintf(f,"%d, %d,%d,%d,",trace[i]->warp_id,trace[i]->sm_id,trace[i]->offset,trace[i]->RoW);//warp_id, pc, RoW
      std::set<uint64_t> coalesced_addr;     
      for (int j=0; j< 32; j++)
      {   uint64_t cache_line_addr = trace[i]->addrs[j]/cache_line_size;
          coalesced_addr.insert(cache_line_addr);
      }
      for (std::set<uint64_t>:: iterator addr=coalesced_addr.begin();addr!=coalesced_addr.end();++addr)
         fprintf(f,"%lld,",*addr);
         fprintf(f,"\n");
     }
     }
     fclose(f);
    }
    */
    free(recv_buffer);
    return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
    recv_thread_started = true;
    channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
    pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    if (recv_thread_started) {
        recv_thread_started = false;
        pthread_join(recv_thread, NULL);
    }
}
