#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>
#include <ctime>
/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* nvbit utility functions */
#include "utils/utils.h"

/* kernel id counter, maintained in system memory */
uint32_t kernel_id = 0;

/* total instruction counter, maintained in system memory, incremented by
 * "counter" every time a kernel completes  */
uint64_t tot_app_instrs = 0;
std::clock_t start;
/* kernel instruction counter, updated by the GPU threads */
__managed__ uint64_t counter = 0;

/* global control variables for this tool */
uint32_t ker_begin_interval = 0;
uint32_t ker_end_interval = UINT32_MAX;
int verbose = 1;
int count_warp_level = 1;
int exclude_pred_off = 0;

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
pthread_mutex_t mutex;

/* instrumentation function that we want to inject, please note the use of
 * 1. "extern "C" __device__ __noinline__" to prevent code elimination by the
 * compiler.
 * 2. NVBIT_EXPORT_FUNC(count_instrs) to notify nvbit the name of the function
 * we want to inject. This name must match exactly the function name */
extern "C" __device__ __noinline__ void count_instrs(int num_instrs,
                                                     int count_warp_level) {
    /* all the active threads will compute the active mask */
    const int active_mask = __ballot(1);
    /* each thread will get a lane id (get_lane_id is in utils/utils.h) */
    const int laneid = get_laneid();
    /* get the id of the first active thread */
    const int first_laneid = __ffs(active_mask) - 1;
    /* count all the active thread */
    const int num_threads = __popc(active_mask);
    /* only the first active thread will perform the atomic */
    if (first_laneid == laneid) {
        if (count_warp_level) {
            atomicAdd((unsigned long long *)&counter, 1 * num_instrs);
        } else {
            atomicAdd((unsigned long long *)&counter, num_threads * num_instrs);
        }
    }
}
NVBIT_EXPORT_FUNC(count_instrs);

extern "C" __device__ __noinline__ void count_pred_off(int predicate,
                                                       int count_warp_level) {
    const int active_mask = __ballot(1);

    const int laneid = get_laneid();

    const int first_laneid = __ffs(active_mask) - 1;

    const int predicate_mask = __ballot(predicate);

    const int mask_off = active_mask ^ predicate_mask;

    const int num_threads_off = __popc(mask_off);
    if (first_laneid == laneid) {
        if (count_warp_level) {
            /* if the predicate mask was off we reduce the count of 1 */
            if (predicate_mask == 0)
                atomicAdd((unsigned long long *)&counter, -1);
        } else {
            atomicAdd((unsigned long long *)&counter, -num_threads_off);
        }
    }
}
NVBIT_EXPORT_FUNC(count_pred_off)

/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We
 * typically do initializations in this call. In this case for instance we get
 * some environment variables values which we use as input arguments to the tool
 */
void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default we instrument everything. */
    GET_VAR_INT(ker_begin_interval, "KERNEL_BEGIN", 0,
                "Beginning of the kernel launch interval where to apply "
                "instrumentation");
    GET_VAR_INT(
        ker_end_interval, "KERNEL_END", UINT32_MAX,
        "End of the kernel launch interval where to apply instrumentation");
    GET_VAR_INT(count_warp_level, "COUNT_WARP_LEVEL", 1,
                "Count warp level or thread level instructions");
    GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 0,
                "Exclude predicated off instruction from count");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 1, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());
}

/* nvbit_at_function_first_load() is executed every time a function is loaded
 * for the first time. Inside this call-back we typically get the vector of SASS
 * instructions composing the loaded hipFunction_t. We can iterate on this vector
 * and insert call to instrumentation functions before or after each one of
 * them. */
void nvbit_at_function_first_load(hipCtx_t ctx, hipFunction_t func) {
    /* Get the static control flow graph of instruction */
    const CFG_t &cfg = nvbit_get_CFG(ctx, func);
    if (cfg.is_degenerate) {
        printf(
            "Warning: Function %s is degenerated, we can't compute basic "
            "blocks statically",
            nvbit_get_func_name(ctx, func));
    }

    if (verbose) {
        printf("Function %s\n", nvbit_get_func_name(ctx, func));
        /* print */
        int cnt = 0;
        for (auto &bb : cfg.bbs) {
            printf("Basic block id %d - num instructions %ld\n", cnt++,
                   bb->instrs.size());
            for (auto &i : bb->instrs) {
                i->print(" ");
            }
        }
    }

    if (verbose) {
        printf("inspecting %s - number basic blocks %ld\n",
               nvbit_get_func_name(ctx, func), cfg.bbs.size());
    }

    /* Iterate on basic block and inject the first instruction */
    for (auto &bb : cfg.bbs) {
        Instr *i = bb->instrs[0];
        /* inject device function */
        nvbit_insert_call(i, "count_instrs", IPOINT_BEFORE);
        /* add size of basic block in number of instruction */
        nvbit_add_call_arg_const_val32(i, bb->instrs.size());
        /* add count warp level option */
        nvbit_add_call_arg_const_val32(i, count_warp_level);
        if (verbose) {
            i->print("Inject count_instr before - ");
        }
    }

    if (exclude_pred_off) {
        /* iterate on instructions */
        for (auto i : nvbit_get_instrs(ctx, func)) {
            /* inject only if instruction has predicate */
            if (i->hasPred()) {
                /* inject function */
                nvbit_insert_call(i, "count_pred_off", IPOINT_BEFORE);
                /* add predicate as argument */
                nvbit_add_call_arg_pred_val(i);
                /* add count warp level option */
                nvbit_add_call_arg_const_val32(i, count_warp_level);
                if (verbose) {
                    i->print("Inject count_instr before - ");
                }
            }
        }
    }
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    /* Identify all the possible CUDA launch events */
    if (cbid == API_CUDA_cuLaunch || cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchGrid || cbid == API_CUDA_cuLaunchGridAsync ||
        cbid == API_CUDA_cuLaunchKernel) {
        /* cast params to cuLaunch_params since if we are here we know these are
         * the right parameters type */
        cuLaunch_params *p = (cuLaunch_params *)params;

        if (!is_exit) {
            start=std::clock();//start tracking time to enter a kernel
            /* if we are entering in a kernel launch:
             * 1. Lock the mutex to prevent multiple kernels to run concurrently
             * (overriding the counter) in case the user application does that
             * 2. Select if we want to run the instrumented or original
             * version of the kernel
             * 3. Reset the kernel instruction counter */

            pthread_mutex_lock(&mutex);
            if (kernel_id >= ker_begin_interval &&
                kernel_id < ker_end_interval) {
                nvbit_enable_instrumented(ctx, p->f, true);
            } else {
                nvbit_enable_instrumented(ctx, p->f, false);
            }
            counter = 0;
        } else {
            double duration = (std::clock()-start)/(double)CLOCKS_PER_SEC;//tracking how long to execute a kernel
            printf("duration:%f\n",duration);
            /* if we are exiting a kernel launch:
             * 1. Wait until the kernel is completed using
             * hipDeviceSynchronize()
             * 2. Get number of thread blocks in the kernel
             * 3. Print the thread instruction counters
             * 4. Release the lock*/
            CUDA_SAFECALL(hipDeviceSynchronize());
            tot_app_instrs += counter;
            int num_ctas = 0;
            if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
                cbid == API_CUDA_cuLaunchKernel) {
                cuLaunchKernel_params *p2 = (cuLaunchKernel_params *)params;
                num_ctas = p2->gridDimX * p2->gridDimY * p2->gridDimZ;
            }
            printf(
                "kernel %d - %s - #thread-blocks %d,  kernel "
                "instructions %ld, total instructions %ld\n",
                kernel_id++, nvbit_get_func_name(ctx, p->f), num_ctas, counter,
                tot_app_instrs);
            pthread_mutex_unlock(&mutex);
        }
    }
}
